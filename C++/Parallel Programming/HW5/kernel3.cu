#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float x0, float y0, float dx, float dy, int width, int* img, int maxIterations, int pitch, int pixelsPerThread) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    for(int j = 0; j < pixelsPerThread; j++){

    
        int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * pixelsPerThread + j;
        int thisY = blockIdx.y * blockDim.y + threadIdx.y;
        float x = x0 + thisX * dx;
        float y = y0 + thisY * dy;
        float tmpX = x, tmpY = y;
        int i;
        for (i = 0; i < maxIterations; ++i)
        {

            if (tmpX * tmpX + tmpY * tmpY > 4.f)
            break;

            float new_x = tmpX * tmpX - tmpY * tmpY;
            float new_y = 2.f * tmpX * tmpY;
            tmpX = x + new_x;
            tmpY = y + new_y;
        }
            

        int* row =  (int*)((char*)img + thisY * pitch);
        row[thisX] = i;
    }
       
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int *d_img, *h_img;
    int pixelsPerThread = 10;
    size_t pitch;
    hipHostAlloc((void **)&h_img, resX * resY * sizeof(int), hipHostMallocDefault);
    hipMallocPitch((void**) &d_img, &pitch, resX * sizeof(int), resY);
    dim3 threadPerBlock(16, 16);
    dim3 numBlock(resX / (threadPerBlock.x *pixelsPerThread), resY / threadPerBlock.y);
    mandelKernel<<<numBlock, threadPerBlock >>>(lowerX, lowerY, stepX, stepY, resX, d_img, maxIterations, pitch, pixelsPerThread);
    hipMemcpy2D(h_img, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_img, resX * resY * sizeof(int));
    hipHostFree(h_img);
    hipFree(d_img);
}
