#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <cmath>
#include <stdlib.h>
#include<sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

//-----------------------DO NOT CHANGE NAMES, ONLY MODIFY VALUES--------------------------------------------

//Final Values that will be compared for correctness
//You may change the function prototypes and definitions, but you need to present final results in these arrays
//-----------------------------Structures for correctness check-------------------
int **SA_Final_student;
int **L_counts_student;
char *L_student;
int F_counts_student[]={0,0,0,0};
//--------------------------------------------------------------------------------

//----------------------------------------------------------------------------------------------------------


//-----------------------DO NOT CHANGE--------------------------------------------

int read_count = 0;
int read_length = 0;

int **SA_Final;
int **L_counts;
char *L;
int F_counts[]={0,0,0,0};


//Read file to get reads
char** inputReads(char *file_path, int *read_count, int *length){
    FILE *read_file = fopen(file_path, "r");
    int ch, lines=0;
    char **reads;
    do                                                                                                 
    {                                                                                                  
        ch = fgetc(read_file);                                                                            
        if (ch == '\n')                                                                                
            lines++;                                                                                   
    } while (ch != EOF);
    rewind(read_file);
    reads=(char**)malloc(lines*sizeof(char*));
    *read_count = lines;
    int i = 0;                                                                                         
    size_t len = 0;                                                                                    
    for(i = 0; i < lines; i++)                                                                         
    {
        reads[i] = NULL;
        len = 0;                                                                                
        getline(&reads[i], &len, read_file);
    }                                                                                                  
    fclose(read_file);
    int j=0;
    while(reads[0][j]!='\n')
        j++;
    *length = j+1;
    for(i=0;i<lines;i++)
        reads[i][j]='$';
    return reads;
}


//Check correctness of values
int checker(){
    int correct = 1;
    for(int i=0; i<read_count*read_length;i++){
        if(L_student[i]!=L[i])
            correct = 0;
        for(int j=0;j<2;j++){
            if(SA_Final_student[i][j]!=SA_Final[i][j])
                correct = 0;
        }
        for(int j=0;j<4;j++){
            if(L_counts_student[i][j]!=L_counts[i][j])
                correct = 0;
        }
    }
    for(int i=0;i<4;i++){
        if(F_counts_student[i]!=F_counts[i])
            correct = 0;
    }
    return correct;
}

//Rotate read by 1 character
void rotateRead(char *read, char *rotatedRead, int length){
    for(int i=0;i<length-1;i++)
        rotatedRead[i]=read[i+1];
    rotatedRead[length-1]=read[0];
}


//Generate Sufixes and their SA's for a read
char** generateSuffixes(char *read, int length, int read_id){
    char **suffixes=(char**)malloc(length*sizeof(char*));
    suffixes[0]=(char*)malloc(length*sizeof(char));
    for(int j=0;j<length;j++)
        suffixes[0][j]=read[j];
    for(int i=1;i<length;i++){
        suffixes[i]=(char*)malloc(length*sizeof(char));
        rotateRead(suffixes[i-1], suffixes[i], length);
    }
    return suffixes;
}

__global__ void rotateReadStudent2D(char* read, char* suffixe, int length, int pitch){
    // for(int i=0;i<length-1;i++)
    //     rotatedRead[i]=read[i+1];
    // rotatedRead[length-1]=read[0];
    // printf("done\n");
    // for(int i=1;i<length;i++){//the suffixes depend on pervious suffixes cannot parallel
    //     suffixes[i]=(char*)malloc(length*sizeof(char));
    //     int thisX = threadIdx.x;
    //     // printf("%d\n", thisX);
    //     rotatedRead[thisX]=read[thisX+1];
    //     rotatedRead[length-1]=read[0];
        
    // }
    
    //TODO make read become share memory
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    extern __shared__ char s_read[];
    s_read[i] = read[i];
    __syncthreads();
    char* row =  (char*)((char*)suffixe + j * pitch);
    // printf("%c %c\n",row[thisX], read[(thisX+thisY)%length]);
    row[i] = s_read[(i+j)%length];

    
}

//TODO
//Kernel function
char* generateSuffixesGPU2D(char *read, int length, int read_i){
    char *suffixes, *d_suffixes, *d_read;
    // suffixes = (char*)malloc(length * length * sizeof(char));
    hipHostAlloc((void **)&suffixes, length * length * sizeof(char), hipHostMallocDefault);
    size_t pitch;
    hipMallocPitch((void**)&d_suffixes, &pitch, length * sizeof(char), length);
    // cudaMalloc((void**)&d_suffixes, length * length * sizeof(char));
    hipMalloc((void**)&d_read, length * sizeof(char));
    hipMemcpy(d_read, read, length * sizeof(char), hipMemcpyHostToDevice);

    // dim3 threadsPerBlock(16, 16);
    // dim3 numBlock(length/threadsPerBlock.x, length/threadsPerBlock.y);
    
    //for small.txt setting
    dim3 threadsPerBlock(9, 9);
    dim3 numBlock(1, 1);

    rotateReadStudent2D<<<numBlock, threadsPerBlock, length*sizeof(char)>>>(d_read, d_suffixes, length, pitch);
    // rotateReadStudent<<<numBlock, threadsPerBlock>>>(d_read, d_suffixes, length);
    
    hipMemcpy2D(suffixes, length * sizeof(char), d_suffixes, pitch, length * sizeof(char), length, hipMemcpyDeviceToHost);

    // for(int i = 0; i < length; i++){
    //     for(int j = 0; j < length; j++){
    //         printf("%c", suffixes[i*length + j]);
    //     }
    //     printf("\n");
    // }
    hipFree(d_suffixes);
    
    //1D version
    // char **suffixes=(char**)malloc(length*sizeof(char*));
    // suffixes[0]=(char*)malloc(length*sizeof(char));
    // printf("inner loop = %d\n", read_i);
    // char *d_suffixes1, *d_suffixes2;
    // cudaMalloc((void**)&d_suffixes1, length * sizeof(char));
    // cudaMalloc((void**)&d_suffixes2, length * sizeof(char));
    // for(int j=0;j<length;j++)
    //     suffixes[0][j]=read[j];
    // for(int i=1;i<length;i++){//the suffixes depend on pervious suffixes cannot parallel
    //     suffixes[i]=(char*)malloc(length*sizeof(char));
    //     cudaMemcpy(d_suffixes1, suffixes[i-1], length * sizeof(char), cudaMemcpyHostToDevice);//suffixes[i-1] to device
    //     // dim3 threadsPerBlock(length, length);
    //     // dim3 numBlock(1, 1);
    //     rotateReadStudent<<<1, length-1>>>(d_suffixes1, d_suffixes2, length);
    //     cudaMemcpy(suffixes[i], d_suffixes2, length * sizeof(char), cudaMemcpyDeviceToHost);//device's suffixes[i] to suffixes[i]
    //     // for(int j = 0; j < length; j++){
    //     //     printf("%c", suffixes[i][j]);
    //     // }
    //     // printf("\n");
        
    // }
    // cudaFree(d_suffixes1);
    // cudaFree(d_suffixes2);
    // char **tmp;
    return suffixes;
    
    
}


__global__ void rotateReadStudent3D(char* read, char* suffixe, int length, int count){
    
    //TODO make read become share memory
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    extern __shared__ char s_read[];
    s_read[i * length + j] = read[i * length + j];
    __syncthreads();
    suffixe[i*length*length + j * length + k] = read[i * length + (j+k)%length];

}


char* generateSuffixesGPU3D(char **read, int length, int count){
    char *suffixes, *d_suffixes, *d_read, *h_read;
    
    int suffixesSize = count * length * length * sizeof(char);
    int readSize = count * length * sizeof(char);
    hipHostAlloc((void **)&suffixes, suffixesSize, hipHostMallocDefault);
    // suffixes = (char*)malloc(size * sizeof(char));
    h_read = (char*)malloc(readSize);
    for(int i = 0; i < count; i++){
        memcpy(h_read + i *length, read[i], length);
    }
    hipMalloc((void**)&d_suffixes, suffixesSize);
    hipMalloc((void**)&d_read, readSize);
    hipMemcpy(d_read, h_read, count*length,hipMemcpyHostToDevice);
    // for(int i = 0; i < count; i++){
    //     cudaMemcpy(d_read + i * length, read[i], length, cudaMemcpyHostToDevice);
    // }
    
    // dim3 threadsPerBlock(32, 32, 32);
    // dim3 numBlock(count/threadsPerBlock.x, length/threadsPerBlock.y, length/threadsPerBlock.z);

    //for small.txt setting
    dim3 threadsPerBlock(128, 128, 128);
    dim3 numBlock(count/threadsPerBlock.x, length/threadsPerBlock.y, length/threadsPerBlock.z);
    // dim3 numBlock(1, 1);

    rotateReadStudent3D<<<numBlock, threadsPerBlock, readSize>>>(d_read, d_suffixes, length, count);
    hipMemcpy(suffixes, d_suffixes, suffixesSize, hipMemcpyDeviceToHost);
    free(h_read);
    hipFree(d_suffixes);
    return suffixes;
}



//Comparator for Suffixes
int compSuffixes(char *suffix1, char *suffix2, int length){
    int ret = 0;
    for(int i=0;i<length;i++){
        if(suffix1[i]>suffix2[i])
            return 1;
        else if(suffix1[i]<suffix2[i])
            return -1;
    }
    return ret;
}


//Calculates the final FM-Index
int** makeFMIndex(char ***suffixes, int read_count, int read_length, int F_count[], char *L){
    int i, j;

    SA_Final=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++)
        SA_Final[i]=(int*)malloc(2*sizeof(int));

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(read_length*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final[i*read_length+j][0]=j;
            SA_Final[i*read_length+j][1]=i;
        }
    }
    
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final[j][0];
                SA_Final[j][0]=SA_Final[j+1][0];
                SA_Final[j+1][0]=temp_int;
                temp_int = SA_Final[j][1];
                SA_Final[j][1]=SA_Final[j+1][1];
                SA_Final[j+1][1]=temp_int;
            }
        }
    }

    free(temp);
    char this_F = '$';
    j=0;
    
    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
    
    //Calculation of L's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }

    return L_count;
}

//-----------------------DO NOT CHANGE--------------------------------------------

int main(int argc, char *argv[]){

    char **reads = inputReads(argv[1], &read_count, &read_length);//Input reads from file
    char ***suffixes=(char***)malloc(read_count*sizeof(char**));//Storage for read-wise suffixes
        
    //-----------------------------Structures for correctness check----------------------------------------------
    L=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    //-----------------------------Structures for correctness check----------------------------------------------
    
    //-----------Default implementation----------------
    //-----------Time capture start--------------------
    struct timeval  TimeValue_Start;
    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Start;
    struct timezone TimeZone_Final;
    long time_start, time_end;
    double time_overhead_default, time_overhead_student;

    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    // printf("REF:\n");
    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);

        // for(int j = 0; j < read_length;j++){
        //     for(int k = 0; k < read_length; k++)
        //         printf("%c", suffixes[i][j][k]);
            
        //     printf("\n");
        // }
        

    }
    
    //Calculate finl FM-Index
    //TODO need to reopen
    // L_counts = makeFMIndex(suffixes, read_count, read_length, F_counts, L);
    
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_default = (time_end - time_start)/1000000.0;
    printf("Start:%ld\tEnd:%ld\tRef:%lf\n", time_start, time_end, time_overhead_default);
    //------------Time capture end----------------------
    //--------------------------------------------------
    
    //-----------Your implementations------------------
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    //-----------Call your functions here--------------------
    // char *suffixes_Student=(char*)malloc(read_count * read_length * read_count * sizeof(char));
    char *suffixes_Student;
    
    
    // for(int i=0;i<read_count;i++){
    //     // char **suffixes_Student_2D;
    //     suffixes_Student[i] = generateSuffixesGPU2D(reads[i], read_length, i);
        
    // }
    
    suffixes_Student = generateSuffixesGPU3D(reads, read_length, read_count);
    gettimeofday(&TimeValue_Final, &TimeZone_Final);

    //print result
    // for(int i = 0; i < 1; i++){
    //     for(int j = 0; j < read_length; j++){
    //         for(int k = 0; k < read_length; k++){
    //             printf("%c", suffixes_Student[i*read_length*read_length + j * read_length + k]);
    //         }
    //         printf("\n");
    //     }
    //     printf("-----------\n");
    // }
    
    //-----------Call your functions here--------------------
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_student = (time_end - time_start)/1000000.0;
    printf("Start:%ld\tEnd:%ld\tStudent:%lf\n", time_start, time_end, time_overhead_student);
    //--------------------------------------------------


    //----------------For debug purpose only-----------------
    //for(int i=0;i<read_count*read_length;i++)        
    //    cout<<L[i]<<"\t"<<SA_Final[i][0]<<","<<SA_Final[i][1]<<"\t"<<L_counts[i][0]<<","<<L_counts[i][1]<<","<<L_counts[i][2]<<","<<L_counts[i][3]<<endl;
    //--------------------------------------------------

    //---------------Correction check and speedup calculation----------------------
    float speedup=0.0;
    // if(checker()==1)
    //    speedup = time_overhead_default/time_overhead_student;
    speedup = time_overhead_default/time_overhead_student;
    cout<<"Speedup="<<speedup<<endl;
    //-----------------------------------------------------------------------------
    return 0;
}
