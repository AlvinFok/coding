#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float x0, float y0, float dx, float dy, int width, int* img, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = x0 + thisX * dx;
    float y = y0 + thisY * dy;
    float tmpX = x, tmpY = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {

        if (tmpX * tmpX + tmpY * tmpY > 4.f)
        break;

        float new_x = tmpX * tmpX - tmpY * tmpY;
        float new_y = 2.f * tmpX * tmpY;
        tmpX = x + new_x;
        tmpY = y + new_y;
    }
        

    int index = (thisY * width + thisX);
    img[index] = i;
       
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int *h_img, *d_img;
    h_img = (int *)malloc(resX * resY * sizeof(int));
    hipMalloc((void**) &d_img, resX * resY * sizeof(int));
    dim3 threadPerBlock(16, 16);
    dim3 numBlock(resX / threadPerBlock.x, resY / threadPerBlock.y);
    mandelKernel<<<numBlock, threadPerBlock >>>(lowerX, lowerY, stepX, stepY, resX, d_img, maxIterations);
    hipMemcpy(h_img, d_img, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img, h_img, resX * resY * sizeof(int));
    hipFree(d_img);
    free(h_img);
}
